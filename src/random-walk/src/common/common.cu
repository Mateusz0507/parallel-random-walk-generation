#include "common/common.cuh"


bool cuda_check_error(hipError_t error, const char* file, const int line, bool terminate)
{
	if (error != hipSuccess)
	{
		std::cerr << "CUDA error: " << hipGetErrorString(error) << ",  occurred in file: "
			<< file << ", line: " << line << "." << std::endl;
		if (terminate)
			exit(EXIT_FAILURE);
		return false;
	}
	return true;
}
