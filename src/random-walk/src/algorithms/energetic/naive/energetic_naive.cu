#include "hip/hip_runtime.h"
#include "algorithms/energetic/naive/energetic_naive.cuh"


__host__ __device__ real_t distance(vector3 p1, vector3 p2)
{
    real_t x_distance = p1.x - p2.x;
    real_t y_distance = p1.y - p2.y;
    real_t z_distance = p1.z - p2.z;
    return sqrt(x_distance * x_distance + y_distance * y_distance + z_distance * z_distance);
}

__host__ __device__ real_t vector_length(real_t x, real_t y, real_t z)
{
    return sqrt(x * x + y * y + z * z);
}

__global__ void iteration(vector3* particles, const int N)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= N)
        return;

    real_t spring_strength = 0.5;
    real_t force_strength = 0.5;
    real_t movement_x = 0, movement_y = 0, movement_z = 0;
    real_t defect, direction_x, direction_y, direction_z, length;

    if (i != 0)
    {
        defect = distance(particles[i], particles[i - 1]) - 1;
        direction_x = particles[i].x - particles[i - 1].x;
        direction_y = particles[i].y - particles[i - 1].y;
        direction_z = particles[i].z - particles[i - 1].z;
        length = vector_length(direction_x, direction_y, direction_z);
        if (length > 0)
        {
            movement_x -= direction_x / length * defect * spring_strength;
            movement_y -= direction_y / length * defect * spring_strength;
            movement_z -= direction_z / length * defect * spring_strength;
        }
    }

    if (i != N - 1)
    {
        defect = distance(particles[i], particles[i + 1]) - 1;
        direction_x = particles[i].x - particles[i + 1].x;
        direction_y = particles[i].y - particles[i + 1].y;
        direction_z = particles[i].z - particles[i + 1].z;
        length = vector_length(direction_x, direction_y, direction_z);
        if (length > 0)
        {
            movement_x -= direction_x / length * defect * spring_strength;
            movement_y -= direction_y / length * defect * spring_strength;
            movement_z -= direction_z / length * defect * spring_strength;
        }
    }

    for (int j = 0; j < N; j++)
    {
        if (j < i - 1 || j > i + 1)
        {
            defect = distance(particles[i], particles[j]) - 1;
            if (defect < 0)
            {
                direction_x = particles[i].x - particles[j].x;
                direction_y = particles[i].y - particles[j].y;
                direction_z = particles[i].z - particles[j].z;
                length = vector_length(direction_x, direction_y, direction_z);
                if (length > 0)
                {
                    movement_x -= direction_x / length * defect * force_strength;
                    movement_y -= direction_y / length * defect * force_strength;
                    movement_z -= direction_z / length * defect * force_strength;
                }
            }
        }
    }

    particles[i].x += movement_x;
    particles[i].y += movement_y;
    particles[i].z += movement_z;
}

bool algorithms::energetic::naive_method::run(vector3** result, int N)
{
    if (allocate_memory(N))
    {
        if (!algorithms::directional_randomization::generate_starting_points(dev_points, N, 1, 5))
            return false;


        /* Create pdb file with points position before the start of the algorithm */
        vector3* points_before_algorithm = new vector3[N];
        if (!cuda_check_continue(hipMemcpy(points_before_algorithm, dev_points, N * sizeof(vector3), hipMemcpyDeviceToHost)))
        {
            release_memory();
            return false;
        }
        create_pdb_file(points_before_algorithm, N, "before");
        open_chimera("before");


        while (!validator.validate(dev_points, N, DISTANCE, EN_PRECISION))
        {
            iteration<<<N/32 + 1, 32>>>(dev_points, N);
        }

        if (!cuda_check_continue(hipMemcpy(*result, dev_points, N * sizeof(vector3), hipMemcpyDeviceToHost)))
        {
            release_memory();
            return false;
        }

        release_memory();
        return true;
    }

    return false;
}

bool algorithms::energetic::naive_method::allocate_memory(int N)
{
    if (N < 0)
        return false;

    if (!cuda_check_continue(hipMalloc(&dev_points, N * sizeof(vector3))))
    {
        dev_points = nullptr;
        return false;
    }

    return true;
}

void algorithms::energetic::naive_method::release_memory()
{
    if (dev_points)
    {
        cuda_check_terminate(hipFree(dev_points));
        dev_points = nullptr;
    }
}

algorithms::energetic::naive_method::naive_method(validators::abstract_validator& validator) : validator{ validator }
{
    this->dev_points = nullptr;
}
