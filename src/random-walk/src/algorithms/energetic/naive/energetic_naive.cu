#include "hip/hip_runtime.h"
#include "algorithms/energetic/naive/energetic_naive.cuh"


__host__ __device__ float distance(algorithms::model::particle p1, algorithms::model::particle p2)
{
    float x_distance = p1.x - p2.x;
    float y_distance = p1.y - p2.y;
    float z_distance = p1.z - p2.z;
    return sqrt(x_distance * x_distance + y_distance * y_distance + z_distance * z_distance);
}

__host__ __device__ float vector_length(float x, float y, float z)
{
    return sqrt(x * x + y * y + z * z);
}

__global__ void iteration(algorithms::model::particle* particles, const int N)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= N)
        return;

    float spring_strength = 0.5;
    float force_strength = 0.5;
    float movement_x = 0, movement_y = 0, movement_z = 0;
    float defect, direction_x, direction_y, direction_z, length;

    if (i != 0)
    {
        defect = distance(particles[i], particles[i - 1]) - 1;
        direction_x = particles[i].x - particles[i - 1].x;
        direction_y = particles[i].y - particles[i - 1].y;
        direction_z = particles[i].z - particles[i - 1].z;
        length = vector_length(direction_x, direction_y, direction_z);
        if (length > 0)
        {
            movement_x -= direction_x / length * defect * spring_strength;
            movement_y -= direction_y / length * defect * spring_strength;
            movement_z -= direction_z / length * defect * spring_strength;
        }
    }

    if (i != N - 1)
    {
        defect = distance(particles[i], particles[i + 1]) - 1;
        direction_x = particles[i].x - particles[i + 1].x;
        direction_y = particles[i].y - particles[i + 1].y;
        direction_z = particles[i].z - particles[i + 1].z;
        length = vector_length(direction_x, direction_y, direction_z);
        if (length > 0)
        {
            movement_x -= direction_x / length * defect * spring_strength;
            movement_y -= direction_y / length * defect * spring_strength;
            movement_z -= direction_z / length * defect * spring_strength;
        }
    }

    for (int j = 0; j < N; j++)
    {
        if (j < i - 1 || j > i + 1)
        {
            defect = distance(particles[i], particles[j]) - 1;
            if (defect < 0)
            {
                direction_x = particles[i].x - particles[j].x;
                direction_y = particles[i].y - particles[j].y;
                direction_z = particles[i].z - particles[j].z;
                length = vector_length(direction_x, direction_y, direction_z);
                if (length > 0)
                {
                    movement_x -= direction_x / length * defect * force_strength;
                    movement_y -= direction_y / length * defect * force_strength;
                    movement_z -= direction_z / length * defect * force_strength;
                }
            }
        }
    }

    particles[i].x += movement_x;
    particles[i].y += movement_y;
    particles[i].z += movement_z;
}

bool algorithms::energetic::naive_method::run(algorithms::model::particle** result, int N)
{
    if (allocate_memory(N))
    {
        if (!algorithms::directional_randomization::generate_starting_points(dev_points, N, 1, 5))
            return false;


        /* Create pdb file with points position before the start of the algorithm */
        algorithms::model::particle* points_before_algorithm = new algorithms::model::particle[N];
        if (!cuda_check_continue(hipMemcpy(points_before_algorithm, dev_points, N * sizeof(model::particle), hipMemcpyDeviceToHost)))
        {
            release_memory();
            return false;
        }
        create_pdb_file(points_before_algorithm, N, "before");
        open_chimera("before");


        while (!validator.validate(dev_points, N, DISTANCE, EN_PRECISION))
        {
            iteration<<<N/32 + 1, 32>>>(dev_points, N);
        }

        if (!cuda_check_continue(hipMemcpy(*result, dev_points, N * sizeof(model::particle), hipMemcpyDeviceToHost)))
        {
            release_memory();
            return false;
        }

        release_memory();
        return true;
    }

    return false;
}

bool algorithms::energetic::naive_method::allocate_memory(int N)
{
    if (N < 0)
        return false;

    if (!cuda_check_continue(hipMalloc(&dev_points, N * sizeof(model::particle))))
    {
        dev_points = nullptr;
        return false;
    }

    return true;
}

void algorithms::energetic::naive_method::release_memory()
{
    if (dev_points)
    {
        cuda_check_terminate(hipFree(dev_points));
        dev_points = nullptr;
    }
}

algorithms::energetic::naive_method::naive_method(validators::abstract_validator& validator) : validator{ validator }
{
    this->dev_points = nullptr;
}
