#include "hip/hip_runtime.h"
#include "algorithms/energetic/naive/energetic_naive.cuh"


__host__ __device__ float distance(algorithms::model::particle p1, algorithms::model::particle p2)
{
    float x_distance = p1.x - p2.x;
    float y_distance = p1.y - p2.y;
    float z_distance = p1.z - p2.z;
    return sqrt(x_distance * x_distance + y_distance * y_distance + z_distance * z_distance);
}

__host__ __device__ float vector_length(float x, float y, float z)
{
    return sqrt(x * x + y * y + z * z);
}

__global__ void iteration(algorithms::model::particle* particles, const int N)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= N)
        return;

    float spring_strength = 0.5;
    float force_strength = 0.5;
    float movement_x = 0, movement_y = 0, movement_z = 0;
    float defect, direction_x, direction_y, direction_z, length;

    if (i != 0)
    {
        defect = distance(particles[i], particles[i - 1]) - 1;
        direction_x = particles[i].x - particles[i - 1].x;
        direction_y = particles[i].y - particles[i - 1].y;
        direction_z = particles[i].z - particles[i - 1].z;
        length = vector_length(direction_x, direction_y, direction_z);
        if (length > 0)
        {
            movement_x -= direction_x / length * defect * spring_strength;
            movement_y -= direction_y / length * defect * spring_strength;
            movement_z -= direction_z / length * defect * spring_strength;
        }
    }

    if (i != N - 1)
    {
        defect = distance(particles[i], particles[i + 1]) - 1;
        direction_x = particles[i].x - particles[i + 1].x;
        direction_y = particles[i].y - particles[i + 1].y;
        direction_z = particles[i].z - particles[i + 1].z;
        length = vector_length(direction_x, direction_y, direction_z);
        if (length > 0)
        {
            movement_x -= direction_x / length * defect * spring_strength;
            movement_y -= direction_y / length * defect * spring_strength;
            movement_z -= direction_z / length * defect * spring_strength;
        }
    }

    for (int j = 0; j < N; j++)
    {
        if (j < i - 1 || j > i + 1)
        {
            defect = distance(particles[i], particles[j]) - 1;
            if (defect < 0)
            {
                direction_x = particles[i].x - particles[j].x;
                direction_y = particles[i].y - particles[j].y;
                direction_z = particles[i].z - particles[j].z;
                length = vector_length(direction_x, direction_y, direction_z);
                if (length > 0)
                {
                    movement_x -= direction_x / length * defect * force_strength;
                    movement_y -= direction_y / length * defect * force_strength;
                    movement_z -= direction_z / length * defect * force_strength;
                }
            }
        }
    }

    particles[i].x += movement_x;
    particles[i].y += movement_y;
    particles[i].z += movement_z;
}

bool algorithms::energetic::naive_method::run(algorithms::model::particle** result, int N)
{
    if (allocate_memory(N))
    {
        generate_random_starting_points(N);


        /* Create pdb file with points position before the start of the algorithm */
        algorithms::model::particle* points_before_algorithm = new algorithms::model::particle[N];
        if (!cuda_check_continue(hipMemcpy(points_before_algorithm, dev_points, N * sizeof(model::particle), hipMemcpyDeviceToHost)))
        {
            release_memory();
            return false;
        }
        create_pdb_file(points_before_algorithm, N, "before");
        open_chimera("before");


        while (!validator.validate(dev_points, N, DISTANCE, EN_PRECISION))
        {
            iteration<<<N/32 + 1, 32>>>(dev_points, N);
        }

        if (!cuda_check_continue(hipMemcpy(*result, dev_points, N * sizeof(model::particle), hipMemcpyDeviceToHost)))
        {
            release_memory();
            return false;
        }

        release_memory();
        return true;
    }

    return false;
}

bool algorithms::energetic::naive_method::allocate_memory(int N)
{
    if (N < 0)
        return false;

    if (!cuda_check_continue(hipMalloc(&dev_points, N * sizeof(model::particle))))
    {
        dev_points = nullptr;
        return false;
    }

    return true;
}

void algorithms::energetic::naive_method::release_memory()
{
    if (dev_points)
    {
        cuda_check_terminate(hipFree(dev_points));
        dev_points = nullptr;
    }
}

bool algorithms::energetic::naive_method::generate_random_starting_points(int N)
{
    model::particle* starting_points = new model::particle[N];

    hiprandState* dev_states = nullptr;
    if (!cuda_check_continue(hipMalloc(&dev_states, N * sizeof(hiprandState))))
    {
        dev_states = nullptr;
        return false;
    }

    model::particle* dev_unit_vectors = nullptr;
    if (!cuda_check_continue(hipMalloc(&dev_unit_vectors, N * sizeof(model::particle))))
    {
        dev_unit_vectors = nullptr;
        return false;
    }

    /* Generate starting points */

    int number_of_blocks = (N + EN_BLOCK_SIZE - 1) / EN_BLOCK_SIZE;
    algorithms::directional_randomization::kernel_setup << <number_of_blocks, EN_BLOCK_SIZE >> > (dev_states, N, std::time(nullptr), 0);
    cuda_check_terminate(hipDeviceSynchronize());

    algorithms::directional_randomization::kernel_generate_random_unit_vectors << <number_of_blocks, EN_BLOCK_SIZE >> > (dev_unit_vectors, dev_states, N);
    model::particle init = { 0.0, 0.0, 0.0 };

    // thrust no operator matches error resolved here https://stackoverflow.com/questions/18123407/cuda-thrust-reduction-with-double2-arrays
    // eventually thrust does not implement operator+ for float3 or double3
    thrust::device_ptr<model::particle> dev_unit_vectors_ptr = thrust::device_ptr<model::particle>(dev_unit_vectors);
    thrust::device_ptr<model::particle> dev_points_ptr = thrust::device_ptr<model::particle>(dev_points);
    model::add_particles add;
    cuda_check_errors_status_terminate(thrust::exclusive_scan(dev_unit_vectors_ptr, dev_unit_vectors_ptr + N, dev_points_ptr, init, add));

    if (dev_unit_vectors)
    {
        cuda_check_terminate(hipFree(dev_unit_vectors));
        dev_unit_vectors = nullptr;
    }
    if (dev_states)
    {
        cuda_check_terminate(hipFree(dev_states));
        dev_states = nullptr;
    }

    return true;
}

algorithms::energetic::naive_method::naive_method(validators::abstract_validator& validator) : validator{ validator }
{
    this->dev_points = nullptr;
}
