#include "hip/hip_runtime.h"
#pragma once
#include "algorithms/model/directional_randomization.cuh"


bool algorithms::directional_randomization::generate_starting_positions(
    vector3* dev_unit_vectors_argument,
    vector3* dev_points_argument,
    const int N,
    const int directional_parametr,
    const int number_of_segments,
    const int seed)
{
    /* Case when number of segments is too small */
    if (number_of_segments < 1)
        return false;

    /* Case when number of segments is too big */
    if (directional_parametr > 0 && N - 1 < number_of_segments)
        return false;

    int number_of_blocks = (N + EN_BLOCK_SIZE - 1) / EN_BLOCK_SIZE;


    hiprandState* dev_states = nullptr;
    if (!cuda_check_continue(hipMalloc(&dev_states, (N - 1) * sizeof(hiprandState))))
    {
        dev_states = nullptr;
        return false;
    }

    algorithms::directional_randomization::kernel_setup <<<number_of_blocks, EN_BLOCK_SIZE>>>(dev_states, N - 1, seed);
    cuda_check_terminate(hipDeviceSynchronize());

    vector3* dev_points = nullptr;
    if (!cuda_check_continue(hipMalloc(&dev_points, N * sizeof(vector3))))
    {
        dev_points = nullptr;
        return false;
    }

    vector3* dev_unit_vectors = nullptr;
    if (!cuda_check_continue(hipMalloc(&dev_unit_vectors, (N - 1) * sizeof(vector3))))
    {
        dev_unit_vectors = nullptr;
        return false;
    }

    matrix* dev_segments_directions_matrices = nullptr;
    if (!cuda_check_continue(hipMalloc(&dev_segments_directions_matrices, number_of_segments * sizeof(matrix))))
    {
        dev_segments_directions_matrices = nullptr;
        return false;
    }


    kernel_generate_segments_directions <<<number_of_blocks, EN_BLOCK_SIZE>>>
        (dev_segments_directions_matrices, dev_states, number_of_segments, seed);
    
    kernel_generate_random_unit_vectors <<<number_of_blocks, EN_BLOCK_SIZE>>>
        (dev_unit_vectors, dev_states, dev_segments_directions_matrices, number_of_segments, N - 1, directional_parametr);


    /*
    * thrust no operator matches error resolved here:
    * https://stackoverflow.com/questions/18123407/cuda-thrust-reduction-with-double2-arrays
    * eventually thrust does not implement operator+ for float3 or double3
    */
    thrust::device_ptr<vector3> dev_unit_vectors_ptr = thrust::device_ptr<vector3>(dev_unit_vectors);
    thrust::device_ptr<vector3> dev_points_ptr = thrust::device_ptr<vector3>(dev_points);
    add_vector3 add;
    cuda_check_errors_status_terminate(thrust::inclusive_scan(dev_unit_vectors_ptr, dev_unit_vectors_ptr + N, dev_points_ptr + 1, add));


    hipMemcpy(dev_unit_vectors_argument, dev_unit_vectors, (N - 1) * sizeof(vector3), hipMemcpyDeviceToDevice);
    hipMemcpy(dev_points_argument, dev_points, N * sizeof(vector3), hipMemcpyDeviceToDevice);


    if (dev_states)
    {
        cuda_check_terminate(hipFree(dev_states));
        dev_states = nullptr;
    }

    if (dev_points)
    {
        cuda_check_terminate(hipFree(dev_points));
        dev_points = nullptr;
    }

    if (dev_unit_vectors)
    {
        cuda_check_terminate(hipFree(dev_unit_vectors));
        dev_unit_vectors = nullptr;
    }

    if (dev_segments_directions_matrices)
    {
        cuda_check_terminate(hipFree(dev_segments_directions_matrices));
        dev_segments_directions_matrices = nullptr;
    }

    return true;
}

__global__ void algorithms::directional_randomization::kernel_setup(
    hiprandState* dev_states, int n, uint64_t seed, uint64_t offset)
{
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    if (index < n)
        hiprand_init(seed, index, offset, &dev_states[index]);
}

__global__ void algorithms::directional_randomization::kernel_generate_segments_directions(
    matrix* dev_segments_directions_matrices, hiprandState* dev_states, int number_of_segments, uint64_t seed)
{
    /*
    * Generation of starting points can be directed towards [1, 0, 0] direction.
    * To change direction towards vector v, you need to change basis of linear space so v is [1, 0, 0] in new basis.
    */
    
    const int tid = threadIdx.x + blockIdx.x * blockDim.x;
    if (tid < number_of_segments)
    {
        real_t alpha = acos(2 * cuda_rand_uniform(&dev_states[tid]) - 1.0);
        real_t beta = cuda_rand_uniform(&dev_states[tid]) * 2 * PI;

        dev_segments_directions_matrices[tid] = matrix(spherical_coordinates(alpha, beta));
    }
}

__global__ void algorithms::directional_randomization::kernel_generate_random_unit_vectors(
    vector3* dev_unit_vectors,
    hiprandState* dev_states,
    matrix* dev_segments_directions_matrices,
    int number_of_segments,
    int n,
    int k)
{
	/*
    * Article that describes uniform distribution on a sphere:
    * https://www.bogotobogo.com/Algorithms/uniform_distribution_sphere.php
	*/

	const int tid = threadIdx.x + blockIdx.x * blockDim.x;
	if (tid < n)
	{
		/* alpha and beta are in [0, pi] */
		real_t alpha = acos(2 * cuda_rand_uniform(&dev_states[tid]) - 1.0);
		real_t beta = cuda_rand_uniform(&dev_states[tid]) * PI;

		for (int i = 0; i < k; i++)
		{
			alpha = acos(M_2_PI * alpha - 1.0);
			beta = acos(M_2_PI * beta - 1.0);
		}

		/* Final beta value is in [0, 2*pi] */
		beta *= 2;

        vector3 v;
        v.x = sin(alpha) * cos(beta);
        v.y = sin(alpha) * sin(beta);
        v.z = cos(alpha);

        int index_of_segment = (tid * number_of_segments) / n;
        dev_unit_vectors[tid] = dev_segments_directions_matrices[index_of_segment].multiply(v);
	}
}
