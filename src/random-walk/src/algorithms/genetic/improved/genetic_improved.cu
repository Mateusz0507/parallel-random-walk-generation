#include "hip/hip_runtime.h"
#include "thrust/scan.h"
#include "thrust/reduce.h"
#include "thrust/device_ptr.h"
#include "thrust/sort.h"
#include "thrust/iterator/zip_iterator.h"

#include "algorithms/genetic/improved/genetic_improved.cuh"
#include "algorithms/model/particle.cuh"
#include "algorithms/genetic/kernels.cuh"

#include <iostream>

enum class datatype
{
	integer,
	vector3
};

void algorithms::genetic::genetic_improved_method::print_device_array(void* dev_ptr, int n, datatype type)
{
	void* host_ptr;
	size_t size;
	switch (type)
	{
	case datatype::integer:
		host_ptr = new int[n];
		size = n * sizeof(int);
		break;
	case datatype::vector3:
		host_ptr = new vector3[n];
		size = n * sizeof(vector3);
		break;
	}
	if (!host_ptr ||
		!cuda_check_continue(hipMemcpy(host_ptr, dev_ptr, size, hipMemcpyDeviceToHost)))
	{
		return;
	}

	for (int i = 0; i < n; i++)
	{
		switch (type)
		{
		case datatype::integer:
			std::cout << ((int*)host_ptr)[i] << std::endl;
			break;
		case datatype::vector3:
			vector3& vec = ((vector3*)host_ptr)[i];
			std::cout << vec.x << ", " << vec.y << ", " << vec.z << std::endl;
			break;
		}
	}
	std::cout << std::endl;
	delete[] host_ptr;
}

bool algorithms::genetic::genetic_improved_method::init(parameters* params)
{
	N1 = params->N - 1;
	generation_size = params->generation_size;
	mutation_ratio = params->mutation_ratio;
	number_of_blocks = (N1 + G_BLOCK_SIZE - 1) / G_BLOCK_SIZE;
	generation_number_of_blocks = (2 * generation_size + G_BLOCK_SIZE - 1) / G_BLOCK_SIZE;

	bool allocation_failure = false;

	fitness = new int[generation_size];
	if (!fitness)
	{
		allocation_failure = true;
		fitness = nullptr;
	}
	new_generation_idx = new int[generation_size];
	if (!allocation_failure && !new_generation_idx)
	{
		allocation_failure = true;
		new_generation_idx = nullptr;
	}

	cuda_allocate((void**)&dev_generation_idx, 2 * params->generation_size * sizeof(int), &allocation_failure);
	cuda_allocate((void**)&dev_fitness, 2 * params->generation_size * sizeof(int), &allocation_failure);
	cuda_allocate((void**)&dev_chromosomes, 2 * (params->N - 1) * params->generation_size * sizeof(vector3), &allocation_failure);
	cuda_allocate((void**)&dev_random_walk, params->N * sizeof(vector3), &allocation_failure);
	cuda_allocate((void**)&dev_states, (params->N - 1) * sizeof(hiprandState), &allocation_failure);
	cuda_allocate((void**)&dev_valid_points, params->N * sizeof(int), &allocation_failure);
	cuda_allocate((void**)&dev_random_walk_idx, params->N * sizeof(int), &allocation_failure);

	if (!allocation_failure)
	{
		// initializing hiprand
		algorithms::randomization::kernel_setup << < number_of_blocks, G_BLOCK_SIZE >> > (dev_states, N1, time(0), 0); // refactor passing arguments
		cuda_check_terminate(hipDeviceSynchronize());

		// initializing cpu random engine
		first_parent_distribution = std::uniform_int_distribution<>(0, generation_size - 1);
		second_parent_distribution = std::uniform_int_distribution<>(0, generation_size - 2 >= 0 ? generation_size - 2 : 0);
		crossover_point_distribution = std::uniform_int_distribution<>(0, N1);

		dev_random_walk_ptr = thrust::device_ptr<vector3>(dev_random_walk);
		for (int i = 0; i < 2 * generation_size; i++)
		{
			dev_chromosomes_ptrs.push_back(thrust::device_ptr<vector3>(dev_chromosomes + i * N1));
		}
		dev_generation_idx_ptr = thrust::device_ptr<int>(dev_generation_idx);
		dev_fitness_ptr = thrust::device_ptr<int>(dev_fitness);
		dev_valid_points_ptr = thrust::device_ptr<int>(dev_valid_points);
		dev_random_walk_idx_ptr = thrust::device_ptr<int>(dev_random_walk_idx);
	}
	else
	{
		terminate();
	}

	return !allocation_failure;
}

algorithms::genetic::genetic_improved_method::genetic_improved_method()
	: rand_device{}, generator{ rand_device() }
{

}

bool algorithms::genetic::genetic_improved_method::run(vector3** particles, void* params)
{
	if (init((parameters*)params))
	{
		int iteration = 0;
		first_generation();
		int solution_idx = -1;
		while (solution_idx < 0)
		{
			next_generation();
			//print_device_array(dev_fitness, 2 * generation_size, datatype::integer);
			compute_fitness_function();
			//print_device_array(dev_fitness, 2 * generation_size, datatype::integer);
			solution_idx = select_population();
			std::cout << "Iteration " << ++iteration << std::endl;
			// print_state();
		}
		copy_solution(particles, solution_idx);
		terminate();
		return true;
	}
	return false;
}

void algorithms::genetic::genetic_improved_method::first_generation()
{
	// generating random walks
	for (int i = 0; i < generation_size; i++)
	{
		algorithms::randomization::kernel_generate_random_unit_vectors << <number_of_blocks, G_BLOCK_SIZE >> > (dev_chromosomes + i * N1, dev_states, N1);
		cuda_check_terminate(hipDeviceSynchronize());
	}

	// initializing idx and fitness function arrays
	kernel_init_tables << <generation_number_of_blocks, G_BLOCK_SIZE >> > (generation_size, dev_generation_idx, dev_fitness);
	cuda_check_terminate(hipDeviceSynchronize());

	// computing fitness function	
	for (int i = 0; i < generation_size; i++)
	{
		fitness_function(i, i);
	}
	cuda_check_terminate(hipMemcpy(dev_fitness, fitness, generation_size * sizeof(int), hipMemcpyHostToDevice));
}

// idx is an index in dev_generation_idx
__global__ void kernel_improved_crossover_and_mutate(vector3* dev_chromosomes, int N1, int child_idx, int first_parent_idx, int second_parent_idx, int* dev_fitness_function, int* dev_generation_idx,
	int generation_size, float mutation_ratio, hiprandState* dev_states)
{
	int tid = threadIdx.x + blockDim.x * blockIdx.x;
	if (tid < N1 && child_idx < 2 * generation_size && first_parent_idx < 2 * generation_size && second_parent_idx < 2 * generation_size)
	{
		int crossover_point = -dev_fitness_function[first_parent_idx];
		dev_chromosomes[tid + dev_generation_idx[child_idx] * N1] =
			dev_chromosomes[tid + dev_generation_idx[tid < crossover_point ? first_parent_idx : second_parent_idx] * N1];

		// mutation
		if (tid == crossover_point || hiprand_uniform(&dev_states[tid]) <= mutation_ratio)
		{
			int i = 0;
			while (i < 15)
			{
				algorithms::randomization::generate_random_unit_vector(&dev_chromosomes[tid + dev_generation_idx[child_idx] * N1], &dev_states[tid]);

			}
		}
	}
}

void algorithms::genetic::genetic_improved_method::next_generation()
{
	for (int i = generation_size; i < 2 * generation_size; i++)
	{
		int first_parent_idx = i % ((generation_size >> 3) > 0 ? (generation_size >> 3) : 1);
		int second_parent_idx = second_parent_distribution(generator);
		if (second_parent_idx >= first_parent_idx)
		{
			second_parent_idx++;
		}
		int crossover_point = crossover_point_distribution(generator);
		kernel_improved_crossover_and_mutate << <number_of_blocks, G_BLOCK_SIZE >> > (dev_chromosomes, N1, i, first_parent_idx, second_parent_idx,
			dev_fitness, dev_generation_idx, generation_size, mutation_ratio, dev_states);
		cuda_check_terminate(hipDeviceSynchronize());
	}
}

void algorithms::genetic::genetic_improved_method::compute_fitness_function()
{
	cuda_check_terminate(hipMemcpy(new_generation_idx, dev_generation_idx + generation_size, generation_size * sizeof(int), hipMemcpyDeviceToHost));
	for (int i = 0; i < generation_size; i++)
	{
		fitness_function(i, new_generation_idx[i]);
	}
	cuda_check_terminate(hipMemcpy(dev_fitness + generation_size, fitness, generation_size * sizeof(int), hipMemcpyHostToDevice));
}

void algorithms::genetic::genetic_improved_method::fitness_function(int fitness_idx, int chromosome_idx)
{
	cuda_check_errors_status_terminate(thrust::exclusive_scan(dev_chromosomes_ptrs[chromosome_idx], dev_chromosomes_ptrs[chromosome_idx] + N1 + 1, dev_random_walk_ptr, init_point, add));

	kernel_improved_fitness_function << <number_of_blocks, G_BLOCK_SIZE >> > (dev_random_walk, N1 + 1, DISTANCE, G_PRECISSION, dev_valid_points, dev_random_walk_idx);
	cuda_check_terminate(hipDeviceSynchronize());

	//{
	//	print_device_array(dev_valid_points, N1 + 1, datatype::integer);
	//	print_device_array(dev_random_walk_idx, N1 + 1, datatype::integer);
	//}

	cuda_check_errors_status_terminate(thrust::sort_by_key(dev_valid_points_ptr, dev_valid_points_ptr + N1 + 1, dev_random_walk_idx_ptr));

	//{
	//	print_device_array(dev_valid_points, N1 + 1, datatype::integer);
	//	print_device_array(dev_random_walk_idx, N1 + 1, datatype::integer);
	//}

	int fitness_function_value;
	cuda_check_terminate(hipMemcpy(&fitness_function_value, dev_random_walk_idx, sizeof(int), hipMemcpyDeviceToHost));

	fitness[fitness_idx] = fitness_function_value ? -fitness_function_value : -N1 - 1;
}

int algorithms::genetic::genetic_improved_method::select_population()
{
	cuda_check_errors_status_terminate(thrust::sort_by_key(dev_fitness_ptr, dev_fitness_ptr + 2 * generation_size, dev_generation_idx_ptr));

	int best_fitness_function;
	cuda_check_terminate(hipMemcpy(&best_fitness_function, dev_fitness, sizeof(int), hipMemcpyDeviceToHost));
	if (best_fitness_function > -N1)
	{
		std::cout << best_fitness_function << std::endl;
		return -1;
	}
	int best_idx;
	cuda_check_terminate(hipMemcpy(&best_idx, dev_generation_idx, sizeof(int), hipMemcpyDeviceToHost));
	return best_idx;
}

void algorithms::genetic::genetic_improved_method::copy_solution(vector3** particles, int idx)
{
	cuda_check_errors_status_terminate(thrust::exclusive_scan(dev_chromosomes_ptrs[idx], dev_chromosomes_ptrs[idx] + N1 + 1, dev_random_walk, init_point, add));
	cuda_check_terminate(hipMemcpy(*particles, dev_random_walk, (N1 + 1) * sizeof(vector3), hipMemcpyDeviceToHost));
}

void algorithms::genetic::genetic_improved_method::terminate()
{
	cuda_release((void**)&dev_fitness);
	cuda_release((void**)&dev_generation_idx);
	cuda_release((void**)&dev_chromosomes);
	cuda_release((void**)&dev_states);
	cuda_release((void**)&dev_random_walk);
	cuda_release((void**)&dev_valid_points);
	cuda_release((void**)&dev_random_walk_idx);
	if (fitness)
	{
		delete[] fitness;
	}
	if (new_generation_idx)
	{
		delete[] new_generation_idx;
	}
}

void algorithms::genetic::genetic_improved_method::print_state()
{
	static int i = 0;
	std::cout << "State " << i++ << std::endl;

	for (int i = 0; i < 2 * generation_size; i++)
	{
		std::cout << "Chromosome " << i << std::endl;
		print_device_array(dev_chromosomes + i * N1, N1, datatype::vector3);
	}
	std::cout << "Fitness function" << std::endl;
	print_device_array(dev_fitness, 2 * generation_size, datatype::integer);
	std::cout << "Random walk" << std::endl;
	print_device_array(dev_random_walk, N1 + 1, datatype::vector3);
	std::cout << "Generation idx" << std::endl;
	print_device_array(dev_generation_idx, 2 * generation_size, datatype::integer);
	std::cout << "Valid points" << std::endl;
	print_device_array(dev_valid_points, N1 + 1, datatype::integer);
}
