#include "hip/hip_runtime.h"
#include "thrust/scan.h"
#include "thrust/reduce.h"
#include "thrust/device_ptr.h"
#include "thrust/sort.h"
#include "thrust/iterator/zip_iterator.h"

#include "algorithms/genetic/naive/genetic.cuh"
#include "algorithms/model/particle.cuh"
#include "algorithms/genetic/kernels.cuh"

#include <iostream>

bool algorithms::genetic::genetic_method::init(parameters* params)
{
	N1 = params->N - 1;
	generation_size = params->generation_size;
	mutation_ratio = params->mutation_ratio;
	number_of_blocks = (N1 + G_BLOCK_SIZE - 1) / G_BLOCK_SIZE;
	generation_number_of_blocks = (2 * generation_size + G_BLOCK_SIZE - 1) / G_BLOCK_SIZE;
	
	bool allocation_failure = false;

	fitness = new int[generation_size];
	if (!fitness) 
	{
		allocation_failure = true;
		fitness = nullptr;
	}
	new_generation_idx = new int[generation_size];
	if(!allocation_failure && !new_generation_idx)
	{
		allocation_failure = true;
		new_generation_idx = nullptr;
	}

	cuda_allocate((void**)&dev_generation_idx, 2 * params->generation_size * sizeof(int), &allocation_failure);
	cuda_allocate((void**)&dev_fitness,	2 * params->generation_size * sizeof(int), &allocation_failure);
	cuda_allocate((void**)&dev_chromosomes, 2 * (params->N - 1)* params->generation_size * sizeof(vector3), &allocation_failure);
	cuda_allocate((void**)&dev_random_walk, params->N * sizeof(vector3), &allocation_failure);
	cuda_allocate((void**)&dev_states, (params->N - 1)* sizeof(hiprandState), &allocation_failure);
	cuda_allocate((void**)&dev_invalid_distances, params->N * sizeof(int), &allocation_failure);

	if (!allocation_failure)
	{
		// initializing hiprand
		algorithms::randomization::kernel_setup<<< number_of_blocks, G_BLOCK_SIZE>>>(dev_states, N1, time(0), 0); // refactor passing arguments
		cuda_check_terminate(hipDeviceSynchronize());

		// initializing cpu random engine
		first_parent_distribution = std::uniform_int_distribution<>(0, generation_size - 1);
		second_parent_distribution = std::uniform_int_distribution<>(0, generation_size - 2 >= 0 ? generation_size - 2 : 0);
		crossover_point_distribution = std::uniform_int_distribution<>(0, N1);

		dev_random_walk_ptr = thrust::device_ptr<vector3>(dev_random_walk);
		for (int i = 0; i < 2 * generation_size; i++)
		{
			dev_chromosomes_ptrs.push_back(thrust::device_ptr<vector3>(dev_chromosomes + i * N1));
		}
		dev_generation_idx_ptr = thrust::device_ptr<int>(dev_generation_idx);
		dev_fitness_ptr = thrust::device_ptr<int>(dev_fitness);
		dev_invalid_distances_ptr = thrust::device_ptr<int>(dev_invalid_distances);
	}
	else
	{
		terminate();
	}

	return !allocation_failure;
}

algorithms::genetic::genetic_method::genetic_method()
	: rand_device{}, generator{rand_device()}
{

}

bool algorithms::genetic::genetic_method::run(vector3** particles, void* params)
{
	if (init((parameters*)params))
	{
		int iteration = 0;
		first_generation();
		int solution_idx = -1;
		while (solution_idx < 0) 
		{
			next_generation();
			compute_fitness_function(); 
			solution_idx = select_population(); 
			std::cout << ++iteration << std::endl;
			//print_state();
 		}
		copy_solution(particles, solution_idx);
		terminate();
		return true;
	}
	return false;
}

void algorithms::genetic::genetic_method::first_generation()
{
	// generating random walks
	for (int i = 0; i < generation_size; i++)
	{
		algorithms::randomization::kernel_generate_random_unit_vectors<<<number_of_blocks, G_BLOCK_SIZE>>>(dev_chromosomes + i * N1, dev_states, N1);
		cuda_check_terminate(hipDeviceSynchronize());
	}

	// initializing idx and fitness function arrays
	kernel_init_tables << <generation_number_of_blocks, G_BLOCK_SIZE >> > (generation_size, dev_generation_idx, dev_fitness);
	cuda_check_terminate(hipDeviceSynchronize());

	// computing fitness function	
	for (int i = 0; i < generation_size; i++)
	{	
		fitness_function(i, i);
	}
	cuda_check_terminate(hipMemcpy(dev_fitness, fitness, generation_size * sizeof(int), hipMemcpyHostToDevice));
}

// idx is an index in dev_generation_idx
__global__ void kernel_crossover_and_mutate(vector3* dev_chromosomes, int N1, int child_idx, int first_parent_idx, int second_parent_idx, int crossover_point, int* dev_generation_idx,
	int generation_size, float mutation_ratio, hiprandState* dev_states)
{
	int tid = threadIdx.x + blockDim.x * blockIdx.x;
	if (tid < N1 && child_idx < 2 * generation_size && first_parent_idx < 2 * generation_size && second_parent_idx < 2 * generation_size) 
	{
		dev_chromosomes[tid + dev_generation_idx[child_idx] * N1] =
			dev_chromosomes[tid + dev_generation_idx[tid < crossover_point ? first_parent_idx : second_parent_idx] * N1];

		// mutation
		if (hiprand_uniform(&dev_states[tid]) <= mutation_ratio)
		{
			algorithms::randomization::generate_random_unit_vector(&dev_chromosomes[tid + dev_generation_idx[child_idx] * N1], &dev_states[tid]);
		}
	}
}

void algorithms::genetic::genetic_method::next_generation()
{
	for (int i = generation_size; i < 2 * generation_size; i++)
	{
		int first_parent_idx = first_parent_distribution(generator);
		int second_parent_idx = second_parent_distribution(generator);
		if (second_parent_idx >= first_parent_idx)
		{
			second_parent_idx++;
		}
		int crossover_point = crossover_point_distribution(generator);
		kernel_crossover_and_mutate << <number_of_blocks, G_BLOCK_SIZE >> > (dev_chromosomes, N1, i, first_parent_idx, second_parent_idx,
			crossover_point, dev_generation_idx, generation_size, mutation_ratio, dev_states);
		cuda_check_terminate(hipDeviceSynchronize());
	}
}

void algorithms::genetic::genetic_method::compute_fitness_function()
{
	cuda_check_terminate(hipMemcpy(new_generation_idx, dev_generation_idx + generation_size, generation_size * sizeof(int), hipMemcpyDeviceToHost));
	for (int i = 0; i < generation_size; i++)
	{
		fitness_function(i, new_generation_idx[i]);
	}
	cuda_check_terminate(hipMemcpy(dev_fitness + generation_size, fitness, generation_size * sizeof(int), hipMemcpyHostToDevice));
}

// TODO: abstract the kernel due to its dual usage
__global__ void kernel_fitness_function(const vector3* dev_data, int N, const real_t distance, const real_t precision, int* dev_invalid_distances)
{
	const int tid = threadIdx.x + blockIdx.x * blockDim.x;
	if (tid < N)
	{
		const int low_range = (N - 1) >> 1;
		const int high_range = N - 1 - low_range;
		const int range = tid < (N >> 1) ? high_range : low_range;

		int invalid_count = 0;

		if (tid + 1 != N && abs(algorithms::model::get_distance(dev_data[tid], dev_data[tid + 1]) - distance) > precision)
		{
			// case when the following vector3 is in different distance than the specified as an parameter
			invalid_count++;
		}

		for (int i = tid + 2, j = i; i < tid + range + 1; i++, j++)
		{
			if (j >= N)
			{
				j -= N;
			}
			if (algorithms::model::get_distance(dev_data[tid], dev_data[j]) < distance - precision)
			{
				invalid_count++;
			}
		}

		dev_invalid_distances[tid] = invalid_count;
	}
}

void algorithms::genetic::genetic_method::fitness_function(int fitness_idx, int chromosome_idx)
{
 	cuda_check_errors_status_terminate(thrust::exclusive_scan(dev_chromosomes_ptrs[chromosome_idx], dev_chromosomes_ptrs[chromosome_idx] + N1 + 1, dev_random_walk_ptr, init_point, add));

	kernel_fitness_function << <number_of_blocks, G_BLOCK_SIZE >> > (dev_random_walk, N1 + 1, DISTANCE, G_PRECISSION, dev_invalid_distances);
	cuda_check_terminate(hipDeviceSynchronize());

	cuda_check_errors_status_terminate(fitness[fitness_idx] = thrust::reduce(dev_invalid_distances_ptr, dev_invalid_distances_ptr + N1));
}

int algorithms::genetic::genetic_method::select_population()
{
 	cuda_check_errors_status_terminate(thrust::sort_by_key(dev_fitness_ptr, dev_fitness_ptr + 2 * generation_size, dev_generation_idx_ptr));

	int best_fitness_function;
	hipMemcpy(&best_fitness_function, dev_fitness, sizeof(int), hipMemcpyDeviceToHost);
	if (best_fitness_function > 0)
	{
		std::cout << best_fitness_function << std::endl;
		return -1;
	}
	int best_idx;
	hipMemcpy(&best_idx, dev_generation_idx, sizeof(int), hipMemcpyDeviceToHost);
	return best_idx;
}

void algorithms::genetic::genetic_method::copy_solution(vector3** particles, int idx)
{
	cuda_check_errors_status_terminate(thrust::exclusive_scan(dev_chromosomes_ptrs[idx], dev_chromosomes_ptrs[idx] + N1 + 1, dev_random_walk, init_point, add));
	cuda_check_terminate(hipMemcpy(*particles, dev_random_walk, (N1 + 1) * sizeof(vector3), hipMemcpyDeviceToHost));
}

void algorithms::genetic::genetic_method::terminate()
{
	cuda_release((void**)&dev_fitness);
	cuda_release((void**)&dev_generation_idx);
	cuda_release((void**)&dev_chromosomes);
	cuda_release((void**)&dev_states);
	cuda_release((void**)&dev_random_walk);
	cuda_release((void**)&dev_invalid_distances);
	if (fitness)
	{
		delete[] fitness;
	}
	if (new_generation_idx)
	{
		delete[] new_generation_idx;
	}
}
