#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <math.h>

struct Particle {
    float x, y, z;
};

hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size);

__global__ void addKernel(int *c, const int *a, const int *b)
{
    int i = threadIdx.x;
    c[i] = a[i] + b[i];
}

void initialize_particles_locations(Particle* particles, const int n)
{
    particles[0].x = particles[0].y = particles[0].z = 0;

    srand((unsigned int)time(NULL));
    for (int i = 1; i < n; i++)
    {
        float versor_x = 2 * ((float)rand() / (float)(RAND_MAX)) - 1;
        float versor_y = 2 * ((float)rand() / (float)(RAND_MAX)) - 1;
        float versor_z = 2 * ((float)rand() / (float)(RAND_MAX)) - 1;
        float versor_length = sqrt(versor_x * versor_x + versor_y * versor_y + versor_z * versor_z);
        versor_x /= versor_length;
        versor_y /= versor_length;
        versor_z /= versor_length;
        particles[i].x = particles[i - 1].x + versor_x;
        particles[i].y = particles[i - 1].y + versor_y;
        particles[i].z = particles[i - 1].z + versor_z;
    }
}

void print_particles(Particle* particles, const int n)
{
    for (int i = 0; i < n; i++)
    {
        float x_distance = particles[i].x - particles[(i - 1 + n) % n].x;
        float y_distance = particles[i].y - particles[(i - 1 + n) % n].y;
        float z_distance = particles[i].z - particles[(i - 1 + n) % n].z;
        float distance_to_last_particle = sqrt(x_distance * x_distance + y_distance * y_distance + z_distance * z_distance);
        printf("Particle %d: x: %f   y: %f   z : %f   distance to last: %f\n",
            i, particles[i].x, particles[i].y, particles[i].z, distance_to_last_particle);
    }
}

__global__ void fix_particles_locations_kernel(Particle* particles, const int n)
{
    int i = threadIdx.x;
    Particle particle = particles[i];

    particle.x = i + 0.6;
    particle.y = i + 0.7;
    particle.z = i + 0.8;

    particles[i] = particle;
}

hipError_t fix_particles_locations(Particle* particles, const int n)
{
    Particle* dev_particles = 0;
    hipError_t cudaStatus;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    // Allocate GPU buffers for three vectors (two input, one output)    .
    cudaStatus = hipMalloc((void**)&dev_particles, n * sizeof(Particle));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(dev_particles, particles, n * sizeof(Particle), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    // Launch a kernel on the GPU with one thread for each element.
    fix_particles_locations_kernel << <1, n >> > (dev_particles, n);

    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "fix_particles_locations launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(particles, dev_particles, n * sizeof(Particle), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

Error:
    hipFree(dev_particles);

    return cudaStatus;
}

int main()
{
    const int arraySize = 5;
    const int a[arraySize] = { 1, 2, 3, 4, 5 };
    const int b[arraySize] = { 10, 20, 30, 40, 50 };
    int c[arraySize] = { 0 };

    const int N = 10;
    Particle particles[N];
    initialize_particles_locations(particles, N);
    print_particles(particles, N);
    fix_particles_locations(particles, N);
    print_particles(particles, N);

    hipError_t cudaStatus = fix_particles_locations(particles, N);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "fix_particles_locations failed!");
        return 1;
    }

    // Add vectors in parallel.
    cudaStatus = addWithCuda(c, a, b, arraySize);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addWithCuda failed!");
        return 1;
    }

    printf("{1,2,3,4,5} + {10,20,30,40,50} = {%d,%d,%d,%d,%d}\n",
        c[0], c[1], c[2], c[3], c[4]);

    // hipDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }

    return 0;
}

// Helper function for using CUDA to add vectors in parallel.
hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size)
{
    int *dev_a = 0;
    int *dev_b = 0;
    int *dev_c = 0;
    hipError_t cudaStatus;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    // Allocate GPU buffers for three vectors (two input, one output)    .
    cudaStatus = hipMalloc((void**)&dev_c, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_a, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_b, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    // Launch a kernel on the GPU with one thread for each element.
    addKernel<<<1, size>>>(dev_c, dev_a, dev_b);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }
    
    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

Error:
    hipFree(dev_c);
    hipFree(dev_a);
    hipFree(dev_b);
    
    return cudaStatus;
}
